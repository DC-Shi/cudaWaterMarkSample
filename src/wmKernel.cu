#include "hip/hip_runtime.h"
#include "wmKernel.cuh"

__global__ void applyKernel(hipfftComplex* array, int width, int height, int startX, int startY)
{
    float maskedValue = 256;
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width*height)
    {
        // This is column-major
        int y = idx % height;
        int x = idx / height;
        // draw a line
        if (100+startX <= x && x <= 120+startX)
            if (40+startY <= y && y <= 180+startY)
            {
                array[idx].x = maskedValue;
            }

        // draw a circle, the (0,0) is left-bottom
        float center1X = 80+startX;
        float center1Y = 80+startY;
        float distance1Sq = (x-center1X)*(x-center1X) + (y-center1Y)*(y-center1Y);

        if (400 <= distance1Sq && distance1Sq <= 1600)
        {
            array[idx].x = maskedValue;
        }

        // draw another circle, the (0,0) is left-bottom
        float center2X = 180+startX;
        float center2Y = 80+startY;
        float distance2Sq = (x-center2X)*(x-center2X) + (y-center2Y)*(y-center2Y);

        if (400 <= distance2Sq && distance2Sq <= 1600)
        {
            float cosTheta = (x-center2X) / sqrt(distance2Sq);
            if (cosTheta < 0.8)
                array[idx].x = maskedValue;
        }

        // draw a line
        if (240+startX <= x && x <= 320+startX)
            if (70+startY <= y && y <= 90+startY)
            {
                array[idx].x = maskedValue;
            }

        // draw another circle, the (0,0) is left-bottom
        float center3X = 380+startX;
        float center3Y = 80+startY;
        float distance3Sq = (x-center3X)*(x-center3X) + (y-center3Y)*(y-center3Y);

        if (400 <= distance3Sq && distance3Sq <= 1600)
        {
            float cosTheta = (x-center3X) / sqrt(distance3Sq);
            float sinTheta = (y-center3Y) / sqrt(distance3Sq);
            if (cosTheta < 0.7 && sinTheta >= 0)
                array[idx].x = maskedValue;
            if (cosTheta > -0.7 && sinTheta <= 0)
                array[idx].x = maskedValue;
            if (340+startX <= x && x <= 420+startX)
                if (70+startY <= y && y <= 90+startY)
                {
                    array[idx].x = maskedValue;
                }
        }
        if (350+startX <= x && x <= 410+startX)
            if (70+startY <= y && y <= 90+startY)
            {
                array[idx].x = maskedValue;
            }
    }
}

void applyKernelToImgAsync(hipfftComplex* array, int width, int height)
{
    size_t threadPerBlock = 1024;
    size_t blocks = (width*height + threadPerBlock - 1)/threadPerBlock;
    // applyKernel<<<blocks, threadPerBlock>>>(array, width, height, -10, -10);
    applyKernel<<<blocks, threadPerBlock>>>(array, width, height, 0, height-200);
    // applyKernel<<<blocks, threadPerBlock>>>(array, width, height, width-500, 0);
    // applyKernel<<<blocks, threadPerBlock>>>(array, width, height, width-500, height-200);
}




__global__ void scaleElement(hipfftComplex* array, int width, int height, float factor)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width * height)
    {
        array[idx].x /= factor;
        array[idx].y /= factor;
    }
}
void scaleComplexAsync(hipfftComplex* array, int width, int height, float factor)
{
    size_t threadPerBlock = 1024;
    size_t blocks = (width*height + threadPerBlock - 1)/threadPerBlock;
    scaleElement<<<blocks, threadPerBlock>>>(array, width, height, factor);
}


void compareTwoImg(ColoredImageType a, ColoredImageType b)
{
    int widthA = FreeImage_GetWidth(a);
    int heightA = FreeImage_GetHeight(a);
    int widthB = FreeImage_GetWidth(b);
    int heightB = FreeImage_GetHeight(b);

    if ((widthA != widthB) || (heightA != heightB))
    {
        std::cout << "Compare result: Two image does not match in size." << std::endl;
        return;
    }
    
    float totalErr = 0;
    float maxErr = 0;

    for (int y = 0; y < heightA; y++)
    {
        for (int x = 0; x < widthA; x++)
        {
            RGBQUAD pixelColorA;
            FreeImage_GetPixelColor(a, x, y, &pixelColorA);
            RGBQUAD pixelColorB;
            FreeImage_GetPixelColor(b, x, y, &pixelColorB);

            auto diffR = abs((float)pixelColorA.rgbRed - pixelColorB.rgbRed);
            auto diffG = abs((float)pixelColorA.rgbGreen - pixelColorB.rgbGreen);
            auto diffB = abs((float)pixelColorA.rgbBlue - pixelColorB.rgbBlue);
            totalErr += diffR;
            totalErr += diffG;
            totalErr += diffB;

            maxErr = max(maxErr, (float)diffR);
            maxErr = max(maxErr, (float)diffG);
            maxErr = max(maxErr, (float)diffB);
        }
    }

    std::cout << "Compare result: Two image max diff " << maxErr << " , average err per pixel " << totalErr/heightA/widthA << std::endl;

}